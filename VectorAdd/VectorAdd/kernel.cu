
#include "hip/hip_runtime.h"


#include <stdio.h>

#define SIZE 1024

// __global__ - kljucna rec - kompajler zna da ce se ova funkcija izvrsavati na gpu
//ova fija ce se izvrsavati u paraleli od strane borjnih threadova

__global__ void vectorAdd(int *a, int *b, int *c, int n){
	
	int i = threadIdx.x;	//jedna nit sabira jedan par
	if (i < n)
		c[i] = a[i] + b[i];
}

int main(){
	/*
	size_t free, total;

	printf("\n");

	cudaMemGetInfo(&free, &total);

	printf("%d KB free of total %d KB\n", free / 1024, total / 1024);
	*/
	/*
	//operisemo sa dva razlicita adresna prostora, jedan je u oviru CPU a drugi u okviru GPU
	//prvo moramo da alociramo prostor na GPU
	//zatim da iskopiramo podatke sa CPU memorije na GPU memoriju (elemente niza a i b)
	//izvrsicemo izracunavanje na GPU (a+b)
	//i nakon toga cemo iskopirati rezultat sa GPU memorje na CPU memoriju (niz c)

	int *a, *b, *c;
	int *d_a, *d_b, *d_c; //d-device
	//imamo razlicite pokazivace za razlicite delove memorije

	a = (int*)malloc(SIZE*sizeof(int));
	b = (int*)malloc(SIZE*sizeof(int));
	c = (int*)malloc(SIZE*sizeof(int));

	cudaMalloc(&d_a, SIZE*sizeof(int));	//alokacije memorije na GPU
	cudaMalloc(&d_b, SIZE*sizeof(int));
	cudaMalloc(&d_c, SIZE*sizeof(int));

	for (int i = 0; i < SIZE; i++){
		a[i] = i;
		b[i] = i;
		c[i] = 0;
	}
	//inicijalizovani podaci
	//nakon alkoacije prostora na GPU kopiramo podatke koje smo upravo inicijalizovali

	cudaMemcpy(d_a, a, SIZE*sizeof(int), cudaMemcpyHostToDevice);
	cudaMemcpy(d_b, b, SIZE*sizeof(int), cudaMemcpyHostToDevice);
	cudaMemcpy(d_c, c, SIZE*sizeof(int), cudaMemcpyHostToDevice);
	
	
	vectorAdd<<<1, SIZE>>>(d_a,d_b,d_c,SIZE);
	//prvi parametar - broj blokova
	//drugi parametar - broj niti u tom bloku


	//copiramo rezultat sa GPU kartice u CPU adresni prosor

	cudaMemcpy(c, d_c, SIZE*sizeof(int), cudaMemcpyDeviceToHost);
	for (int i = 0; i < 10; i++)
		printf("c[%d] = %d \n", i, c[i]);

	free(a);
	free(b);
	free(c);

	//oslobadjamo prostor na GPU
	cudaFree(d_a);
	cudaFree(d_b);
	cudaFree(d_c);
	*/
	return 0; 
}