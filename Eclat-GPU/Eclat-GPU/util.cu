#include "hip/hip_runtime.h"
#include "util.h"

unsigned int FileRead(istream & is, vector <char> & buff) {
	is.read(&buff[0], buff.size());
	return is.gcount();
}

unsigned int CountLines(const vector <char> & buff, int sz) {
	int newlines = 0;
	const char * p = &buff[0];
	for (int i = 0; i < sz; i++) {
		if (p[i] == '\n') {
			newlines++;
		}
	}
	return newlines;
}

int fileLineCnt(char* fileName){
	time_t now = time(0);
	ifstream ifs(fileName);
	int n = 0;
	string s;
	while (getline(ifs, s)) {
		n++;
	}
	cout << "system: File size calulation time[s]: " <<time(0) - now << endl;
	return n;
}


void tokenizeLine(char** line)
{
	int i = 0;
	char* s;
	line[i++] = strtok(line[0], " \t\n");
	while (line[i - 1] != NULL)
	{
		line[i++] = strtok(NULL, " \t\n");
	}
}

void resetTokens(char* line, char** tokens)
{
	tokens[0] = line;
	for (int i = 1; i< 10; tokens[i++] = 0);
}


/*__global__ void loading(int** memMatrix, vector<boost::dynamic_bitset<> > bitMatrix, int numOfTransactions, int numOfItemsInLine){
	
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int num;
	if (idx < numOfTransactions){
		for (int i = 0; i < numOfItemsInLine; i++){

			if (num = memMatrix[idx][i] == 0) break;
			bitMatrix[num][idx] = 1;
		}
	}
}
*/